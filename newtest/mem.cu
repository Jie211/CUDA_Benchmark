#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "MT.h"

texture<int2, 1, hipReadModeElementType> vec_tex;

__device__ __inline__ double shfl_xor(double val, int const lane)
{
  return __hiloint2double(__shfl_xor(__double2hiint(val),lane), __shfl_xor(__double2loint(val),lane));
}

void GetHead(const char *bx, const char *col, const char *ptr, int *n, int *nnz)
{
  FILE *in1, *in2, *in3;

  if((in1 = fopen(bx, "r")) == NULL)
  {
    printf("** error in head %s file open **\n", bx);
    exit(-1);
  }

  if((in2 = fopen(col, "r")) == NULL)
  {
    printf("** error head %s file open **\n", col);
    exit(-1);
  }

  if((in3 = fopen(ptr, "r")) == NULL)
  {
    printf("** error head %s file open **\n", ptr);
    exit(-1);
  }
  int N11, N12, N21, N22, N31, N32;
  int NZ1, NZ2, NZ3;

  fscanf(in1, "%d %d %d\n", &N11, &N12, &NZ1);
  fscanf(in2, "%d %d %d\n", &N21, &N22, &NZ2);
  fscanf(in3, "%d %d %d\n", &N31, &N32, &NZ3);

  if(N11!=N12)
  {
    printf("** error in %s N!=M **\n", bx);
    exit(-1);
  }
  if(N21!=N22)
  {
    printf("** error in %s N!=M **\n", col);
    exit(-1);
  }
  if(N31!=N32)
  {
    printf("** error in %s N!=M **\n", ptr);
    exit(-1);
  }

  if(N11 != N21 || N21!=N31 || N31!=N11)
  {
    printf("** error N was not same in 3files **\n");
    exit(-1);
  }

  if(NZ1 != NZ2 || NZ2!=NZ3 || NZ3!=NZ1)
  {
    printf("** error NNZ was not same in 3files **\n");
    exit(-1);
  }
  *n = N11;
  *nnz = NZ1;

  fclose(in1);
  fclose(in2);
  fclose(in3);
}
void GetData(const char *file1, const char *file2, const char *file3, int *col, int *ptr, double *val, double *b, double *x, int N, int NZ)
{
  FILE *in1,*in2,*in3;
  if((in1 = fopen(file1, "r")) == NULL)
  {
    printf("** error %s file open **", file1);
    exit(0);
  }

  if((in2 = fopen(file2, "r")) == NULL)
  {
    printf("** error %s file open **", file2);
    exit(0);
  }

  if((in3 = fopen(file3, "r")) == NULL)
  {
    printf("** error %s file open **", file3);
    exit(0);
  }
  int getint;
  double getdouble, getdouble2;
  int skip1, skip2, skip3;

  fscanf(in1, "%d %d %d\n", &skip1, &skip2, &skip3);
  fscanf(in2, "%d %d %d\n", &skip1, &skip2, &skip3);
  fscanf(in3, "%d %d %d\n", &skip1, &skip2, &skip3);
  for(int i=0;i<NZ;i++)
  {
    fscanf(in1,"%d %le\n",&getint,&getdouble);
    col[i] = getint;
    val[i] = getdouble;
  }

  for(int i=0;i<N+1;i++)
  {
    fscanf(in2,"%d\n",&getint);
    ptr[i] = getint;
  }

  for(int i=0;i<N;i++)
  {
    fscanf(in3,"%le %le\n",&getdouble,&getdouble2);
    b[i] = getdouble;
    x[i] = getdouble2;
  }


  fclose(in1);
  fclose(in2);
  fclose(in3);
}


double gettimeofday_sec()
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv.tv_sec + (double)tv.tv_usec*1e-6;
}
__global__ void
mv1(int n, double *val, int *col, int *ptr, double *b, double *c)
{
  long int i, j;
  for(i=0;i<n;i++){
    double tmp=0.0;
    for(j=ptr[i];j<ptr[i+1];j++){
      tmp+=val[j]*b[col[j]];
    }
    c[i]=tmp;
  }
  /* __syncthreads(); */
}
__global__ void
mv2(int n, double *val, int *col, int *ptr, double *b, double *c)
{
  long row=blockDim.x * blockIdx.x + threadIdx.x;
  long int i;
  if(row<n){
    double tmp=0.0;
    long int row_start=ptr[row];
    long int row_end=ptr[row+1];
    for(i=row_start;i<row_end;i++){
      tmp+=val[i]*b[col[i]];
    }
    /* __syncthreads(); */
    c[row]=tmp;
    /* printf("%d %.12e\n", row, c[row]); */
  }
  /* __syncthreads(); */
}
__global__ void
mv3(int n, double *val, int *col, int *ptr, double *b, double *c){
  /* extern __shared__ volatile double vals[]; */
  extern __shared__ double vals[];

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    vals[threadIdx.x] = 0.0;

    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      vals[threadIdx.x]+=val[jj] * b[col[jj]];
    }

    if(lane <16)
      vals[threadIdx.x] += vals[threadIdx.x +16];
    if(lane<8)
      vals[threadIdx.x] += vals[threadIdx.x + 8];
    if(lane<4)
      vals[threadIdx.x] += vals[threadIdx.x + 4];
    if(lane<2)
      vals[threadIdx.x] += vals[threadIdx.x + 2];
    if(lane<1)
      vals[threadIdx.x] += vals[threadIdx.x + 1];

    if(lane == 0){
      c[row] += vals[threadIdx.x];
    }
  }
}

__global__ void
mv4(int n, double *val, int *col, int *ptr, double *b, double *c){
  /* extern __shared__ volatile double vals[]; */
  extern __shared__ double vals[];

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int2 v;

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    vals[threadIdx.x] = 0.0;

    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      v=tex1Dfetch(vec_tex, col[jj]);
      /* vals[threadIdx.x]+=val[jj] * b[col[jj]]; */
      vals[threadIdx.x]+=val[jj] * __hiloint2double(v.y, v.x);
    }

    if(lane <16)
      vals[threadIdx.x] += vals[threadIdx.x +16];
    if(lane<8)
      vals[threadIdx.x] += vals[threadIdx.x + 8];
    if(lane<4)
      vals[threadIdx.x] += vals[threadIdx.x + 4];
    if(lane<2)
      vals[threadIdx.x] += vals[threadIdx.x + 2];
    if(lane<1)
      vals[threadIdx.x] += vals[threadIdx.x + 1];

    if(lane == 0){
      c[row] += vals[threadIdx.x];
    }
  }
}

__global__ void
mv5(int n, double *val, int *col, int *ptr, double *b, double *c)
{
  long row=blockDim.x * blockIdx.x + threadIdx.x;
  long int i;
  int2 v;
  if(row<n){
    double tmp=0.0;
    long int row_start=ptr[row];
    long int row_end=ptr[row+1];
    for(i=row_start;i<row_end;i++){
      v=tex1Dfetch(vec_tex, col[i]);
      /* tmp+=val[i]*b[col[i]]; */
      tmp+=val[i]*__hiloint2double(v.y, v.x);
    }
    /* __syncthreads(); */
    c[row]=tmp;
    /* printf("%d %.12e\n", row, c[row]); */
  }
  /* __syncthreads(); */
}

__global__ void
mv6(int n, double *val, int *col, int *ptr, const double * __restrict__ b, double *c){
  /* extern __shared__ volatile double vals[]; */
  extern __shared__ double vals[];

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    vals[threadIdx.x] = 0.0;

    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      vals[threadIdx.x]+=val[jj] * b[col[jj]];
    }

    if(lane <16)
      vals[threadIdx.x] += vals[threadIdx.x +16];
    if(lane<8)
      vals[threadIdx.x] += vals[threadIdx.x + 8];
    if(lane<4)
      vals[threadIdx.x] += vals[threadIdx.x + 4];
    if(lane<2)
      vals[threadIdx.x] += vals[threadIdx.x + 2];
    if(lane<1)
      vals[threadIdx.x] += vals[threadIdx.x + 1];

    if(lane == 0){
      c[row] += vals[threadIdx.x];
    }
  }
}

__global__ void
mv7(int n, double *val, int *col, int *ptr, const double * __restrict__ b, double *c){

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    double sum=0.0;
    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      sum+=val[jj] * b[col[jj]];
    }

    sum+=shfl_xor(sum, 16);
    sum+=shfl_xor(sum, 8);
    sum+=shfl_xor(sum, 4);
    sum+=shfl_xor(sum, 2);
    sum+=shfl_xor(sum, 1);

    if(lane == 0){
      c[row] += sum;
    }
  }
}


int main(int argc, char const* argv[])
{
  int i, j, N, NNZ;

  hipSetDeviceFlags(hipDeviceMapHost);

  GetHead(argv[1], argv[2], argv[3], &N, &NNZ);

  printf("n=%d, nnz=%d\n", N, NNZ);
  printf("----------------------------------------------\n");
  double *val, *b, *c;
  int *col, *ptr;

  val=(double *)malloc(sizeof(double)*NNZ);
  col=(int *)malloc(sizeof(int)*NNZ);
  ptr=(int *)malloc(sizeof(int)*(N+1));
  b=(double *)malloc(sizeof(double)*N);
  c=(double *)malloc(sizeof(double)*N);

  double *dval, *db, *dc;
  int *dcol, *dptr;

  GetData(argv[1], argv[2], argv[3], col, ptr, val, b, c, N, NNZ);

  for(i=0;i<N;i++){
    b[i]=genrand_real3();
    c[i]=0.0;
  }
  //------------------------------------
  double st1, et1, t1, sum1=0.0;
  st1=gettimeofday_sec();
  for(i=0;i<N;i++){
    double tmp=0.0;
    for(j=ptr[i];j<ptr[i+1];j++){
      tmp+=val[j] * b[col[j]];
    }
    c[i]=tmp;
  }
  et1=gettimeofday_sec();
  t1=et1-st1;
  for(i=0;i<N;i++){
    sum1+=c[i];
    c[i]=0.0;
  }
  printf("sum1=%f,t1=%.12e:CPU\n",sum1,t1);

  //------------------------------------
  omp_set_num_threads(8);

  double st2, et2, t2, sum2=0.0;
  st2=gettimeofday_sec();
  double tmp_omp=0.0;
  
 #pragma omp parallel for private(j) reduction(+:tmp_omp) schedule(static) firstprivate(c, val, b) lastprivate(c)
  for(i=0;i<N;i++){
    tmp_omp=0.0;
    for(j=ptr[i];j<ptr[i+1];j++){
      tmp_omp+=val[j] * b[col[j]];
    }
    c[i]=tmp_omp;
  }
  et2=gettimeofday_sec();
  t2=et2-st2;
  for(i=0;i<N;i++){
    sum2+=c[i];
    c[i]=0.0;
  }
  printf("sum2=%f,t2=%.12e:OpenMP\n",sum2,t2);
  //------------------------------------
  checkCudaErrors( hipMalloc((void **)&dval, sizeof(double)*NNZ) );
  checkCudaErrors( hipMalloc((void **)&dcol, sizeof(int)*NNZ) );
  checkCudaErrors( hipMalloc((void **)&dptr, sizeof(int)*(N+1)) );
  checkCudaErrors( hipMalloc((void **)&db, sizeof(double)*N) );
  checkCudaErrors( hipMalloc((void **)&dc, sizeof(double)*N) );

  double st3, et3, t3, sum3=0.0;
  st3=gettimeofday_sec();
  checkCudaErrors( hipMemcpy(dval, val, sizeof(double)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dcol, col, sizeof(int)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dptr, ptr, sizeof(int)*(N+1), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(db, b, sizeof(double)*N, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemset(dc, 0, sizeof(double)*N) );

  mv1<<<1, 1>>>(N, dval, dcol, dptr, db, dc);

  checkCudaErrors(hipPeekAtLastError());


  checkCudaErrors( hipMemcpy(c, dc, sizeof(double)*N, hipMemcpyDeviceToHost) );

  et3=gettimeofday_sec();

  t3=et3-st3;
  for(i=0;i<N;i++){
    sum3+=c[i];
    c[i]=0.0;
  }
  printf("sum3=%f,t3=%.12e:mv1\n",sum3,t3);


   //------------------------------------
 
  double st4, et4, t4, sum4=0.0;
  st4=gettimeofday_sec();
  checkCudaErrors( hipMemcpy(dval, val, sizeof(double)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dcol, col, sizeof(int)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dptr, ptr, sizeof(int)*(N+1), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(db, b, sizeof(double)*N, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemset(dc, 0, sizeof(double)*N) );
  
  int ThreadPerBlock=960;
  int BlockPerGrid=ceil((double)N/(double)ThreadPerBlock);

  mv2<<<BlockPerGrid, ThreadPerBlock>>>(N, dval, dcol, dptr, db, dc);

  checkCudaErrors(hipPeekAtLastError());


  checkCudaErrors( hipMemcpy(c, dc, sizeof(double)*N, hipMemcpyDeviceToHost) );

  et4=gettimeofday_sec();

  t4=et4-st4;
  for(i=0;i<N;i++){
    sum4+=c[i];
    c[i]=0.0;
  }
  printf("sum4=%f,t4=%.12e:mv2\n",sum4,t4);

 
  //--------------------------------------
  
  double st5, et5, t5, sum5=0.0;
  st5=gettimeofday_sec();
  checkCudaErrors( hipMemcpy(dval, val, sizeof(double)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dcol, col, sizeof(int)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dptr, ptr, sizeof(int)*(N+1), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(db, b, sizeof(double)*N, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemset(dc, 0, sizeof(double)*N) );
  
  ThreadPerBlock=960;
  /* BlockPerGrid=ceil((double)N/(double)ThreadPerBlock/32); */
  BlockPerGrid=(N-1)/(ThreadPerBlock/32)+1;

  mv3<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(N, dval, dcol, dptr, db, dc);

  checkCudaErrors(hipPeekAtLastError());


  checkCudaErrors( hipMemcpy(c, dc, sizeof(double)*N, hipMemcpyDeviceToHost) );

  et5=gettimeofday_sec();

  t5=et5-st5;
  for(i=0;i<N;i++){
    sum5+=c[i];
    c[i]=0.0;
  }
  printf("sum5=%f,t5=%.12e:mv3\n",sum5,t5);

  //--------------------------------------------
  checkCudaErrors( hipHostRegister(val, sizeof(double)*NNZ, hipHostRegisterDefault) );
  checkCudaErrors( hipHostRegister(col, sizeof(int)*NNZ, hipHostRegisterDefault) );
  checkCudaErrors( hipHostRegister(ptr, sizeof(int)*(N+1), hipHostRegisterDefault) );
  checkCudaErrors( hipHostRegister(b, sizeof(double)*N, hipHostRegisterDefault) );
  checkCudaErrors( hipHostRegister(c, sizeof(double)*N, hipHostRegisterDefault) );

  double st6, et6, t6, sum6=0.0;
  st6=gettimeofday_sec();

  checkCudaErrors( hipMemcpy(dval, val, sizeof(double)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dcol, col, sizeof(int)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dptr, ptr, sizeof(int)*(N+1), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(db, b, sizeof(double)*N, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemset(dc, 0, sizeof(double)*N) );
  
  ThreadPerBlock=960;
  BlockPerGrid=(N-1)/(ThreadPerBlock/32)+1;

  mv3<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(N, dval, dcol, dptr, db, dc);

  checkCudaErrors(hipPeekAtLastError());


  checkCudaErrors( hipMemcpy(c, dc, sizeof(double)*N, hipMemcpyDeviceToHost) );

  et6=gettimeofday_sec();

  t6=et6-st6;
  for(i=0;i<N;i++){
    sum6+=c[i];
    c[i]=0.0;
  }
  printf("sum6=%f,t6=%.12e:mv3+pinned memory\n",sum6,t6);
  
  checkCudaErrors( hipHostUnregister(val) );
  checkCudaErrors( hipHostUnregister(col) );
  checkCudaErrors( hipHostUnregister(ptr) );
  checkCudaErrors( hipHostUnregister(b) );
  checkCudaErrors( hipHostUnregister(c) );

  //------------------------------------------
  double *val2, *b2, *c2;
  int *col2, *ptr2;


  checkCudaErrors( hipHostAlloc((void **)&val2, sizeof(double)*NNZ, hipHostMallocMapped)  );
  checkCudaErrors( hipHostAlloc((void **)&col2, sizeof(int)*NNZ, hipHostMallocMapped)  );
  checkCudaErrors( hipHostAlloc((void **)&ptr2, sizeof(int)*(N+1), hipHostMallocMapped)  );
  checkCudaErrors( hipHostAlloc((void **)&b2, sizeof(double)*N, hipHostMallocMapped)  );
  checkCudaErrors( hipHostAlloc((void **)&c2, sizeof(double)*N, hipHostMallocMapped)  );

  double *dval2, *db2, *dc2;
  int *dcol2, *dptr2;

  GetData(argv[1], argv[2], argv[3], col2, ptr2, val2, b2, c2, N, NNZ);

  for(i=0;i<N;i++){
    b2[i]=b[i];
    c2[i]=0.0;
  }

  checkCudaErrors( hipHostGetDevicePointer( (void **)&dval2, (void *)val2, 0) );
  checkCudaErrors( hipHostGetDevicePointer( (void **)&dcol2, (void *)col2, 0) );
  checkCudaErrors( hipHostGetDevicePointer( (void **)&dptr2, (void *)ptr2, 0) );
  checkCudaErrors( hipHostGetDevicePointer( (void **)&db2, (void *)b2, 0) );
  checkCudaErrors( hipHostGetDevicePointer( (void **)&dc2, (void *)c2, 0) );

  double st7, et7, t7, sum7=0.0;
  st7=gettimeofday_sec();

  ThreadPerBlock=960;
  BlockPerGrid=(N-1)/(ThreadPerBlock/32)+1;

  mv3<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(N, dval2, dcol2, dptr2, db2, dc2);

  checkCudaErrors(hipPeekAtLastError());

  checkCudaErrors(hipDeviceSynchronize());

  et7=gettimeofday_sec();

  t7=et7-st7;
  for(i=0;i<N;i++){
    sum7+=c2[i];
    c[i]=0.0;
  }
  printf("sum7=%f,t7=%.12e:mv3+mapped memory\n",sum7,t7);
  
  checkCudaErrors( hipHostFree(val2) );
  checkCudaErrors( hipHostFree(col2) );
  checkCudaErrors( hipHostFree(ptr2) );
  checkCudaErrors( hipHostFree(b2) );
  checkCudaErrors( hipHostFree(c2) );

  //------------------------------------------
  double *val3, *b3, *c3;
  int *col3, *ptr3;

  checkCudaErrors( hipMallocManaged((void **)&val3, sizeof(double)*NNZ) );
  checkCudaErrors( hipMallocManaged((void **)&col3, sizeof(int)*NNZ) );
  checkCudaErrors( hipMallocManaged((void **)&ptr3, sizeof(int)*(N+1)) );
  checkCudaErrors( hipMallocManaged((void **)&b3, sizeof(double)*N)  );
  checkCudaErrors( hipMallocManaged((void **)&c3, sizeof(double)*N)  );


  GetData(argv[1], argv[2], argv[3], col3, ptr3, val3, b3, c3, N, NNZ);

  for(i=0;i<N;i++){
    b3[i]=b[i];
    c3[i]=0.0;
  }

  double st8, et8, t8, sum8=0.0;
  st8=gettimeofday_sec();

  ThreadPerBlock=960;
  BlockPerGrid=(N-1)/(ThreadPerBlock/32)+1;
  /* BlockPerGrid=(N-1)/(ThreadPerBlock)+1; */

  mv3<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(N, val3, col3, ptr3, b3, c3);

  /* mv2<<<BlockPerGrid, ThreadPerBlock>>>(N, val3, col3, ptr3, b3, c3); */

  checkCudaErrors(hipPeekAtLastError());

  checkCudaErrors(hipDeviceSynchronize());

  et8=gettimeofday_sec();

  t8=et8-st8;
  for(i=0;i<N;i++){
    sum8+=c3[i];
    /* c[i]=0.0; */
  }
  printf("sum8=%f,t8=%.12e:mv3+unifield memory\n",sum8,t8);


  checkCudaErrors( hipFree(val3) );
  checkCudaErrors( hipFree(col3) );
  checkCudaErrors( hipFree(ptr3) );
  checkCudaErrors( hipFree(b3) );
  checkCudaErrors( hipFree(c3) );


//--------------------------------------

  
  double st9, et9, t9, sum9=0.0;
  st9=gettimeofday_sec();
  checkCudaErrors( hipMemcpy(dval, val, sizeof(double)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dcol, col, sizeof(int)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dptr, ptr, sizeof(int)*(N+1), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(db, b, sizeof(double)*N, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemset(dc, 0, sizeof(double)*N) );
  
  ThreadPerBlock=960;
  /* BlockPerGrid=ceil((double)N/(double)ThreadPerBlock); */
  /* BlockPerGrid=N; */
  BlockPerGrid=(N-1)/(ThreadPerBlock/32)+1;

  checkCudaErrors( hipBindTexture(NULL, vec_tex, db, sizeof(double)*N) );
  mv4<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(N, dval, dcol, dptr, db, dc);
  checkCudaErrors( hipUnbindTexture(vec_tex) );

  checkCudaErrors(hipPeekAtLastError());

  checkCudaErrors( hipMemcpy(c, dc, sizeof(double)*N, hipMemcpyDeviceToHost) );

  et9=gettimeofday_sec();

  t9=et9-st9;
  for(i=0;i<N;i++){
    sum9+=c[i];
    c[i]=0.0;
  }
  printf("sum9=%f,t9=%.12e:mv3+texture memory\n",sum9,t9);
//--------------------------------------

  
  double st10, et10, t10, sum10=0.0;
  st10=gettimeofday_sec();
  checkCudaErrors( hipMemcpy(dval, val, sizeof(double)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dcol, col, sizeof(int)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dptr, ptr, sizeof(int)*(N+1), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(db, b, sizeof(double)*N, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemset(dc, 0, sizeof(double)*N) );
  
  ThreadPerBlock=960;
  BlockPerGrid=ceil((double)N/(double)ThreadPerBlock);
  /* BlockPerGrid=N; */
  /* BlockPerGrid=(N-1)/(ThreadPerBlock/32)+1; */

  checkCudaErrors( hipBindTexture(NULL, vec_tex, db, sizeof(double)*N) );
  mv5<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(N, dval, dcol, dptr, db, dc);
  checkCudaErrors( hipUnbindTexture(vec_tex) );

  checkCudaErrors(hipPeekAtLastError());

  checkCudaErrors( hipMemcpy(c, dc, sizeof(double)*N, hipMemcpyDeviceToHost) );

  et10=gettimeofday_sec();

  t10=et10-st10;
  for(i=0;i<N;i++){
    sum10+=c[i];
    c[i]=0.0;
  }
  printf("sum10=%f,t10=%.12e:mv2+texture memory\n",sum10,t10);

  //--------------------------------------
  double st11, et11, t11, sum11=0.0;
  st11=gettimeofday_sec();
  checkCudaErrors( hipMemcpy(dval, val, sizeof(double)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dcol, col, sizeof(int)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dptr, ptr, sizeof(int)*(N+1), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(db, b, sizeof(double)*N, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemset(dc, 0, sizeof(double)*N) );
  
  ThreadPerBlock=128;
  /* BlockPerGrid=ceil((double)N/(double)ThreadPerBlock); */
  /* BlockPerGrid=N; */
  BlockPerGrid=(N-1)/(ThreadPerBlock/32)+1;

  mv6<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(N, dval, dcol, dptr, db, dc);

  checkCudaErrors(hipPeekAtLastError());

  checkCudaErrors( hipMemcpy(c, dc, sizeof(double)*N, hipMemcpyDeviceToHost) );

  et11=gettimeofday_sec();

  t11=et11-st11;

  for(i=0;i<N;i++){
    sum11+=c[i];
    c[i]=0.0;
  }
  printf("sum11=%f,t11=%.12e:mv3+readonly memory\n",sum11,t11);


  //--------------------------------------
  
  double st12, et12, t12, sum12=0.0;
  st12=gettimeofday_sec();
  checkCudaErrors( hipMemcpy(dval, val, sizeof(double)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dcol, col, sizeof(int)*NNZ, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(dptr, ptr, sizeof(int)*(N+1), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(db, b, sizeof(double)*N, hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemset(dc, 0, sizeof(double)*N) );
  
  ThreadPerBlock=128;
  /* BlockPerGrid=ceil((double)N/(double)ThreadPerBlock); */
  /* BlockPerGrid=N; */
  BlockPerGrid=(N-1)/(ThreadPerBlock/32)+1;

  mv7<<<BlockPerGrid, ThreadPerBlock>>>(N, dval, dcol, dptr, db, dc);

  checkCudaErrors(hipPeekAtLastError());

  checkCudaErrors( hipMemcpy(c, dc, sizeof(double)*N, hipMemcpyDeviceToHost) );

  et12=gettimeofday_sec();

  t12=et12-st12;
  for(i=0;i<N;i++){
    sum12+=c[i];
    c[i]=0.0;
  }
  printf("sum12=%f,t12=%.12e:mv3-shared memory+readonly memory+warp_shuffle\n",sum12,t12);


  checkCudaErrors( hipFree(dval) );
  checkCudaErrors( hipFree(dcol) );
  checkCudaErrors( hipFree(dptr) );
  checkCudaErrors( hipFree(db) );
  checkCudaErrors( hipFree(dc) );




  free(val);
  free(col);
  free(ptr);
  free(b);
  free(c);
  return 0;
}
